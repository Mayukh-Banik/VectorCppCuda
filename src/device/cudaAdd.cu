#include "hip/hip_runtime.h"
#include "add.h"
#include <hip/hip_runtime.h>
#include <iostream>


__global__ void add_kernel(int a, int b, int* c)
{
    *c = a + b;
}

void add_GPU(int a, int b, int* c)
{

    int* d_c;
    hipError_t err;
    err = hipMalloc((void **)&d_c, sizeof(int));
    if (err != hipSuccess) 
    {
        std::cerr << "Failed to allocate device memory" << hipGetErrorString(err) << std::endl;
        exit(EXIT_FAILURE);
    }
    add_kernel<<<1, 1>>>(a, b, d_c);
    err = hipPeekAtLastError();
    if (err != hipSuccess) 
    {
        std::cerr << "Kernel launch failed" <<  hipGetErrorString(err) << std::endl;
        hipFree(d_c); 
        exit(EXIT_FAILURE);
    }
    err = hipDeviceSynchronize();
    if (err != hipSuccess) 
    {
        std::cerr << "Kernel execution failed" << hipGetErrorString(err) << std::endl;
        hipFree(d_c); 
        exit(EXIT_FAILURE);
    }
    err = hipMemcpy(c, d_c, sizeof(int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) 
    {
        std::cerr << "Failed to copy data from device to host" << hipGetErrorString(err) << std::endl;
        hipFree(d_c);  
        exit(EXIT_FAILURE);
    }
    hipFree(d_c);
    std::cerr << "GPU" << std::endl;
}

